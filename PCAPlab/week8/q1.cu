#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#define N 1024
__global__ void wordFreq(char *sent,char *word,int n,int *d_count)
{	
	int i = threadIdx.x;
	for(int j=0; j<n; j++) 
	{		
		if(sent[i+j]!=word[j]) return;
	}
	atomicAdd(d_count,1);
}

int main(void)
{
	char sent[N],word[100],*d_sent,*d_word;
	int count=0, *d_count;
	printf("Enter a sentence: "); fgets(sent,N,stdin);
	printf("Enter a word: "); scanf("%s",word);
	printf("Sentence: %sWord: %s\n",sent,word);
	hipMalloc((void **)&d_sent,strlen(sent)*sizeof(char));
	hipMalloc((void **)&d_word,strlen(word)*sizeof(char));
	hipMalloc((void **)&d_count,sizeof(int));
	hipMemcpy(d_sent,sent,strlen(sent)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_word,word,strlen(word)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int), hipMemcpyHostToDevice);
	wordFreq<<<1,strlen(sent)-strlen(word)>>>(d_sent,d_word,strlen(word), d_count);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	printf("Word occurences: %d\n",count);
	hipFree(d_sent);
	hipFree(d_word);
	hipFree(d_count);
	return 0;
}