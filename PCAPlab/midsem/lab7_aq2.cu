
#include <hip/hip_runtime.h>
//Selection sort on matrix rows. Each row is sorted by one thread.

__global__ void ssortmatr(int *A, int size)
{
    int rowid = threadIdx.x;
    for(int i=0; i<size-1; i++)
    {
        int mini=i;
        for(int j=i+1; j<size; j++)
            if(A[rowid*3+mini]>A[rowid*3+j]) mini=j;
        if(mini!=i)
        {
            int temp = A[rowid*3+mini];
            A[rowid*3+mini]=A[rowid*3+i];
            A[rowid*3+i]=temp;
        }
    }
}