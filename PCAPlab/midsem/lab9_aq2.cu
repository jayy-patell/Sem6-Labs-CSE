#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
_global_ void replaceElements(int *mat,int *B,int m,int n){
  int rid=blockIdx.y*blockDim.y+threadIdx.y;
  int cid=blockIdx.x*blockDim.x+threadIdx.x;
  if(rid<m&&cid<n){
    if(rid==cid)
      B[rid*n+cid]=0;
    else if(rid>cid){
      int sd=0;
      int temp=mat[rid*n+cid];
      while(temp>0){
        sd+=(temp%10);
        temp/=10;
      }
      B[rid*n+cid]=sd;
    }
    else{
      int fact=1;
      int num=mat[rid*n+cid];
      while(num>0){
        fact*=num;
        num--;
      }
      B[rid*n+cid]=fact;
    }
  }
}
int main(){
  int *A,*B,m,n;
  int *d_A,*d_B;
  printf("Enter dimensions: ");
  scanf("%d%d",&m,&n);
  printf("Enter the elements: ");
  A=(int*)malloc(m*n*sizeof(int));
  B=(int*)malloc(m*n*sizeof(int));
  for(int i=0;i<m*n;i++)
    scanf("%d",&A[i]);
  printf("A:-");
  for(int i=0;i<m*n;i++){
    if(i%n==0)
      printf("\n%d ",A[i]);
    else
      printf("%d ",A[i]);
  }
  printf("\nB:-");
  hipMalloc((void**)&d_A,m*n*sizeof(int));
  hipMalloc((void**)&d_B,m*n*sizeof(int));
  hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
  dim3 grid(ceil(n/32.0),ceil(m/32.0),1);
  dim3 block(32,32,1);
  replaceElements<<<grid,block>>>(d_A,d_B,m,n);
  hipMemcpy(B,d_B,m*n*sizeof(int),hipMemcpyDeviceToHost);
  for(int i=0;i<m*n;i++){
    if(i%n==0)
      printf("\n%d ",B[i]);
    else
      printf("%d ",B[i]);
  }
  printf("\n");
  hipFree(d_A);
  hipFree(d_B);
  return 0;
}