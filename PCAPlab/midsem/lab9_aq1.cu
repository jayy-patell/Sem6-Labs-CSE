%%cu

#include<stdio.h>
#include "hip/hip_runtime.h"
#include "device_launch_paramters.h"

_gobal_ void kernel(int *A,int *B,int n,int m)
{
    int r = threadIdx.x;
    int c = threadIdx.y;
    int rsum = 0, csum = 0;
    

    if(A[r*m+c]%2==0)
    {
        for(int i=0;i<m;i++)
        {
            rsum += A[r*m+i];
        }
        B[r*m+c] = rsum;
    }
    else
    {
        for(int i=0;i<n;i++)
        {
            csum += A[i*m+c];
        }
        B[r*m+c] = csum;
    }
}

int main()
{
    int A[12]={1,2,3,4,5,6,7,8,9,10,11,12},B[12];
    int *dA,*dB;
    hipMalloc((void **)&dA,sizeof(int)*4*3);
    hipMalloc((void **)&dB,sizeof(int)*4*3);
    hipMemcpy(dA, A, sizeof(int)*4*3, hipMemcpyHostToDevice);
    dim3 grid(1,1,1);
    dim3 block(4,3,1);
    kenrel<<<grid,block>>>(dA,dB);
    hipMemcpy(B,dB,sizeof(int)*4*3,hipMemcpyDeviceToHost);
    for(int i=0; i<4; i++)
    {
        for(int j=0; j<3; j++) printf("%d ",B[i*3+j]);
        printf("\n");
    }
    hipFree(dA);
    hipFree(dB);
    return 0;
}