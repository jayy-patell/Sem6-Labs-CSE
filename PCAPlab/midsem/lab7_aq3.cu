#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
_global_ void odd(int *A, int size)
{
    int gtid = threadIdx.x;
    if(gtid%2!=0 && ((gtid+1) < size))
    {
        if(A[gtid]>A[gtid+1])
        {
            int temp = A[gtid];
            A[gtid] =A[gtid+1];
            A[gtid+1] = temp;
        }
    }
}
_global_ void even(int *A, int size)
{
    int gtid = threadIdx.x;
    if(gtid%2==0 && ((gtid+1) < size))
    {
        if(A[gtid]>A[gtid+1])
        {
            int temp = A[gtid];
            A[gtid] =A[gtid+1];
            A[gtid+1] = temp;
        }
    }
}
int main()
{
    int size=8;
    int A[size]={23,45,76,12,98,34,9,25};
    int *dA;
    hipMalloc((void **)&dA,sizeof(int)*size);
    hipMemcpy(dA, A, sizeof(int)*size, hipMemcpyHostToDevice);
    for(int i=0; i<size/2; i++)
    {
        odd<<<1,size>>>(dA,size);
        even<<<1,size>>>(dA,size);
    }
    hipMemcpy(A,dA,sizeof(int)*size,hipMemcpyDeviceToHost);
    for(int i=0; i<size; i++)
    {
        printf("%d ",A[i]);
    }
    hipFree(dA);
    return 0;
}