//Q2 and Q3 done in rough notebook.

%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
_global_ void reverse_words(char *S,int *starts, int n)
{
    int i=threadIdx.x;
    int start=starts[i], end;
    end=starts[i+1]-2;
    printf("%d - start - %d - %c ; end - %d - %c\n",i, start,S[start],end,S[end]);
    while(start<end)
    {
        char temp=S[start];
        S[start]=S[end];
        S[end]=temp;
        start++;
        end--;
    }
}
int main()
{
    int n = 4, size=0, starts[n+1], startindex=0, *dstarts;
    char A[100], *dA;
    strcpy(A,"This is my sentence");
    puts(A);
    size=strlen(A);
    starts[startindex++]=0;
    for(int i=0; i<size; i++)
    {
        if(A[i]==' ') starts[startindex++]=i+1;
    }
    starts[startindex]=size+1;
    hipMalloc((void **)&dA,sizeof(char)*size);
    hipMalloc((void *)&dstarts,sizeof(int)(n+1));
    hipMemcpy(dA, A, sizeof(char)*size, hipMemcpyHostToDevice);
    hipMemcpy(dstarts, starts, sizeof(int)*(n+1), hipMemcpyHostToDevice);
    reverse_words<<<1,n>>>(dA,dstarts,size);
    hipMemcpy(A,dA, sizeof(char)*size, hipMemcpyDeviceToHost);
	  puts(A);
    hipFree(dA);
    return 0;
}