#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__constant__ int ks=3;
__constant__ int k[3];

__global__ void oneDconv(int *a,int *r, int as)
{	
	int gtid = blockIdx.x*blockDim.x+threadIdx.x;
	int h = ks/2;
	if (gtid<as) 
	{
		int result = 0, i, ii;
		for(i=0; i<ks; i++)
		{
			ii=gtid-h+i;
			if(ii>=0 && ii<as) result+=a[ii]*k[i];
		}
		r[gtid]=result;
	}
}
int main(void) {
	int a[10]={1,2,3,4,5,6,7,8,9,10},mask[3]={1,2,1},r[10];
	int as=10,ks=3,size1=as*sizeof(int);
	int *da,*dr;
	hipMalloc((void **)&da,size1);
	hipMalloc((void **)&dr,size1);
	hipMemcpy(da,a,size1,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(k),mask,sizeof(int)*ks);
	oneDconv<<<1,10>>>(da,dr,as);
	hipMemcpy(r,dr,size1,hipMemcpyDeviceToHost);
	for(int i=0; i<as; i++) printf("%d ",r[i]);
	hipFree(da);
	hipFree(dr);
	return 0;
}	