#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void matmul(int *a, int *b, int *t,int n)
{
	int r = blockIdx.y*blockDim.y+threadIdx.y;
	int c=blockIdx.x*blockDim.x + threadIdx.x;
	int q=blockDim.x*gridDim.x;
	int sum=0;
	for(int k=0; k<n; k++) sum+=a[r*n+k]*b[k*q+c];
	t[r*q+c]=sum;
}
int main(void)
{
	int *a, *b, *t, m,n,p,q;
	int *d_a,*d_b,*d_t;
	printf("m value: "); scanf("%d",&m);
	printf("n value: "); scanf("%d",&n);
	printf("p value: "); scanf("%d",&p);
	printf("q value: "); scanf("%d",&q);
	int size = sizeof(int)*m*n, size1 = sizeof(int)*p*q, size2=sizeof(int)*m*q;
	a = (int *) malloc(size);
	b = (int *) malloc(size1);
	t = (int *) malloc(size2);
	printf("Enter matrix A: ");
	for(int i=0; i<m*n; i++) scanf("%d",&a[i]);
	printf("Enter matrix B: ");
	for(int i=0; i<p*q; i++) scanf("%d",&b[i]);
	for(int i=0; i<m; i++)
	{
		for(int j=0; j<n; j++)
		{
			printf("%d ",a[i*n+j]);
		}
		printf("\n");
	}
	for(int i=0; i<p; i++)
	{
		for(int j=0; j<q; j++)
		{
			printf("%d ",b[i*q+j]);
		}
		printf("\n");
	}
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size1);
	hipMalloc((void **) &d_t,size2);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size1,hipMemcpyHostToDevice);
	dim3 block(2,2,1), grid(ceil(q/2),ceil(m/2),1);
	matmul<<<grid,block>>>(d_a,d_b,d_t,n);
	hipMemcpy(t,d_t,size2,hipMemcpyDeviceToHost);
	printf("Result vector:\n");
	for(int i=0; i<m; i++)
	{
		for(int j=0; j<q; j++) printf("%d ",t[i*q+j]);
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_t);
	return 0;	
}