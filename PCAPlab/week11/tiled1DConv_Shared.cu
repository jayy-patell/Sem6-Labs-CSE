#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define TW 4
#define MW 5

//N = array, M = mask, P = result

__global__ void tiled1Dconv(int *N, int *M,int *P, int width)
{	
	int i= blockIdx.x*blockDim.x+threadIdx.x,bx=blockIdx.x, tx = threadIdx.x;
	int h = MW/2;
	__shared__ int Ns[TW + MW -1];
	int halo_left_index = (bx -1)*TW + tx;
	if (tx >= TW -h) Ns[tx - (TW -h)] = (halo_left_index < 0) ? 0 : N[halo_left_index];
	Ns[h + tx] = N[i];
	int halo_right_index = (bx + 1)*TW + tx;
	if (tx < h) Ns[h +TW + tx] = (halo_right_index > width) ?  0 : N[halo_right_index];
	for(int j=0; j<8; j++) printf("%d - %d\n",i,Ns[j]);
	__syncthreads();
	int sum=0;
	for(int j=0; j<MW; j++) sum+=Ns[tx + j] * M[j];
	P[i]=sum;
}

int main(void) {
	int width = 16;
	int N[16]={1,2,3,4,5,6,7,8,1,2,3,4,5,6,7,8},M[MW]={1,1,2,1,1},P[16];
	int *dN,*dM, *dP;
	hipMalloc((void **)&dN,sizeof(int)*width);
	hipMalloc((void **)&dP,sizeof(int)*width);
	hipMalloc((void **)&dM,sizeof(int)*MW);
	hipMemcpy(dN,N,sizeof(int)*width,hipMemcpyHostToDevice);
	hipMemcpy(dM,M,sizeof(int)*MW,hipMemcpyHostToDevice);
	tiled1Dconv<<<1,TW>>>(dN,dM,dP,width);
	hipMemcpy(P,dP,sizeof(int)*MW,hipMemcpyDeviceToHost);
	for(int i=0; i<width; i++) printf("%d ",P[i]);
	hipFree(dM);
	hipFree(dN);
	hipFree(dP);
	return 0;
}	