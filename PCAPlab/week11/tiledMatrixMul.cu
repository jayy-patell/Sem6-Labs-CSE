#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#define BW 2
#define TW 2
#define WIDTH 4

__global__ void matmul(int *a, int *b, int *t)
{
	__shared__ int MDs[TW][TW];
	__shared__ int NDs[TW][TW];
	int bx=blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
	int r=by*TW + ty, c = bx*TW+tx;
	int pval=0;
	for(int m=0; m<WIDTH/TW; m++)
	{
		MDs[ty][tx]=a[r*WIDTH + m*TW + tx];
		NDs[ty][tx]=b[(m*TW + ty)*WIDTH + c];
		__syncthreads();
		for(int k=0; k<TW; k++)
		{
			pval+=MDs[ty][k]*NDs[k][tx];
		}
		__syncthreads();
	}
	t[r*WIDTH + c] = pval;
}
int main(void)
{
	int *a, *b, *t;
	int *d_a,*d_b,*d_t;
	int size = sizeof(int)*WIDTH*WIDTH;
	a = (int *) malloc(size);
	b = (int *) malloc(size);
	t = (int *) malloc(size);
	printf("Enter matrix A (4x4): ");
	for(int i=0; i<WIDTH*WIDTH; i++) scanf("%d",&a[i]);
	printf("Enter matrix B: ");
	for(int i=0; i<WIDTH*WIDTH; i++) scanf("%d",&b[i]);
	hipMalloc((void **) &d_a,size);
	hipMalloc((void **) &d_b,size);
	hipMalloc((void **) &d_t,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	int numblocks = WIDTH/BW;
	dim3 block(BW,BW,1), grid(numblocks,numblocks,1);
	matmul<<<grid,block>>>(d_a,d_b,d_t);
	hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
	printf("Result vector:\n");
	for(int i=0; i<WIDTH; i++)
	{
		for(int j=0; j<WIDTH; j++) printf("%d ",t[i*WIDTH+j]);
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_t);
	return 0;	
}